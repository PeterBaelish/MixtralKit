
#include <hip/hip_runtime.h>
// nvcc -shared -o stream_manage.so stream_manage.cu -L/usr/local/cuda/lib64 -lcudart

extern "C" hipStream_t createStream() {
    hipStream_t stream;
    hipStreamCreate(&stream);
    return stream;
}

extern "C" void copyCpuToGpuOnStream(int8_t *dst, const int8_t *src, int n, hipStream_t stream) {
    hipMemcpyAsync(dst, src, n * sizeof(int8_t), hipMemcpyHostToDevice, stream);
}

extern "C" void copy2DTensorCpuToGpuOnStream(int8_t *dst, const int8_t *src, int rows, int cols, hipStream_t stream) {
    size_t size = rows * cols * sizeof(int8_t);
    hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}


extern "C" void synchronizeStream(hipStream_t stream) {
    hipStreamSynchronize(stream);
}

extern "C" void destroyStream(hipStream_t stream) {
    hipStreamDestroy(stream);
}
